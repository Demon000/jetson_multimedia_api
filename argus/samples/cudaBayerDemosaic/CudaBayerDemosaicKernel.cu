#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "CudaBayerDemosaicKernel.h"
#include <stdio.h>

#ifndef CU_EGL_COLOR_FORMAT_BAYER_RGGB
#define CU_EGL_COLOR_FORMAT_BAYER_RGGB (0x2D)
#define CU_EGL_COLOR_FORMAT_BAYER_BGGR (0x2E)
#define CU_EGL_COLOR_FORMAT_BAYER_GRBG (0x2F)
#define CU_EGL_COLOR_FORMAT_BAYER_GBRG (0x30)
#endif

// Constant used to store the component ordering of the Bayer input (used by bayerToRgba).
// These values provide the indexes into the original data that will provide an RGGB ordering.
__constant__ unsigned int bayerPattern[4];

// Converts a 16-bit Bayer quad to 32bit RGBA. The Bayer components are provided
// in the order they're stored in the buffer, as this function will also handle
// the component ordering during conversion using the 'bayerPattern' constant.
static __device__ uchar4
bayerToRgba(unsigned short bayerQuad[4])
{
    // Signed 16-bit Bayer maps 1<<14 to white.
    unsigned int whitePoint = 0xfff;
    unsigned int maxRgba = 0xff;

    // Order the Bayer components based on the format component ordering.
    unsigned int shift = 4;
    unsigned int mask = 0xfff;
    unsigned int r  = (bayerQuad[bayerPattern[0]] >> shift) & mask;
    unsigned int g1 = (bayerQuad[bayerPattern[1]] >> shift) & mask;
    unsigned int g2 = (bayerQuad[bayerPattern[2]] >> shift) & mask;
    unsigned int b  = (bayerQuad[bayerPattern[3]] >> shift) & mask;

    unsigned int rb = b * maxRgba / whitePoint;
    unsigned int rg = (g1 + g2) * maxRgba / whitePoint / 2;
    unsigned int rr = r * maxRgba / whitePoint;

    if (r > whitePoint || g1 > whitePoint || g2 > whitePoint || b > whitePoint ||
        rr > maxRgba || rg > maxRgba || rb > maxRgba) {
        printf("wp: %04x, r: %04x, g1: %04x, g2: %04x, b: %04x, rr: %04x, rg: %04x, rb: %04x\n",
               whitePoint, r, g1, g2, b, rr, rg, rb);
    }

    // Map [0, 1<<14] to [0, 255].
    uchar4 rgba;
    rgba.x = rb;
    rgba.y = rg;
    rgba.z = rr;

    return rgba;
}

// Demosaics a Bayer buffer into an RGBA output.
__global__ void
bayerDemosaicKernel(unsigned short* bayerSrc,
                    int bayerWidth,
                    int bayerHeight,
                    int bayerPitch,
                    uchar4* rgbaDst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int stepX = blockDim.x * gridDim.x;
    int stepY = blockDim.y * gridDim.y;

    int rgbaWidth = bayerWidth / 2;
    int rgbaHeight = bayerHeight / 2;

    for (int col = x; col < rgbaWidth; col += stepX)
    {
        for (int row = y; row < rgbaHeight; row += stepY)
        {
            // Extract the Bayer quad.
            unsigned short* bayerOffset = bayerSrc + (col * 2) + (row * bayerPitch);
            unsigned short bayerQuad[4];
            bayerQuad[0] = *(bayerOffset);
            bayerQuad[1] = *(bayerOffset + 1);
            bayerQuad[2] = *(bayerOffset + (bayerPitch / 2));
            bayerQuad[3] = *(bayerOffset + (bayerPitch / 2) + 1);

            // Demosaic the Bayer quad to RGBA.
            uchar4 rgba = bayerToRgba(bayerQuad);

            // Optional: Add gain (useful to debug dark output).
            unsigned int gain = 1;
            rgba.x = umin(rgba.x * gain, 255);
            rgba.y = umin(rgba.y * gain, 255);
            rgba.z = umin(rgba.z * gain, 255);

            // Output the RGBA pixels to the output buffer.
            rgbaDst[rgbaWidth * row + col] = rgba;
        }
    }
}

// Sets the Bayer pattern constant used to order Bayer components.
static void setBayerPatternConstant(int bayerFormat)
{
    unsigned int pattern[4];
    if (bayerFormat == CU_EGL_COLOR_FORMAT_BAYER_RGGB)
    {
        pattern[0] = 0;
        pattern[1] = 1;
        pattern[2] = 2;
        pattern[3] = 3;
    }
    else if (bayerFormat == CU_EGL_COLOR_FORMAT_BAYER_BGGR)
    {
        pattern[0] = 3;
        pattern[1] = 1;
        pattern[2] = 2;
        pattern[3] = 0;
    }
    else if (bayerFormat == CU_EGL_COLOR_FORMAT_BAYER_GRBG)
    {
        pattern[0] = 1;
        pattern[1] = 0;
        pattern[2] = 3;
        pattern[3] = 2;
    }
    else // bayerFormat == CU_EGL_COLOR_FORMAT_BAYER_GBRG
    {
        pattern[0] = 2;
        pattern[1] = 0;
        pattern[2] = 3;
        pattern[3] = 1;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(bayerPattern), pattern, sizeof(pattern));
}

// Entrypoint to the CUDA Bayer Demosaic.
int cudaBayerDemosaic(hipDeviceptr_t bayerSrc,
                      int bayerWidth,
                      int bayerHeight,
                      int bayerPitch,
                      int bayerFormat,
                      hipDeviceptr_t rgbaDst)
{
    setBayerPatternConstant(bayerFormat);

    dim3 threadsPerBlock(32, 32);
    dim3 blocks(2, 2);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    bayerDemosaicKernel<<<blocks, threadsPerBlock>>>(
            (unsigned short*)bayerSrc, bayerWidth, bayerHeight, bayerPitch, (uchar4*)rgbaDst);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsedMillis;
    hipEventElapsedTime(&elapsedMillis, start, stop);

    printf("CUDA KERNEL:      Processed frame in %fms\n", elapsedMillis);

    return 0;
}
